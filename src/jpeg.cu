#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "dct.h"
#include "quantize.h"
#include "dpcm.h"
#include "rle.h"

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);

inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
                line);
        if(abort) {
            exit(code);
        }
    }
}

#else
#define cudaCheckError(ans) ans
#endif

//cuda image.h
std::shared_ptr<ImageRgb> cudaConvertBytesToImage(std::vector<unsigned char> bytes, unsigned int width, unsigned int height);
std::vector<unsigned char> cudaConvertImageToBytes(std::shared_ptr<ImageRgb> image);

std::shared_ptr<ImageYcbcr> cudaConvertRgbToYcbcr(std::shared_ptr<ImageRgb> input);
std::shared_ptr<ImageRgb> cudaConvertYcbcrToRgb(std::shared_ptr<ImageYcbcr> input);

std::shared_ptr<ImageBlocks> cudaConvertYcbcrToBlocks(std::shared_ptr<ImageYcbcr> input, int block_size);
std::shared_ptr<ImageYcbcr> cudaConvertBlocksToYcbcr(std::shared_ptr<ImageBlocks> input, int block_size);

//cuda dct.h
std::vector<std::shared_ptr<PixelYcbcr>> cudaDCT(std::vector<std::shared_ptr<PixelYcbcr>> pixels, int block_size, bool all);
std::vector<std::shared_ptr<PixelYcbcr>> cudaIDCT(std::vector<std::shared_ptr<PixelYcbcr>> pixels, int block_size, bool all);

//cuda quantize.h
std::vector<std::shared_ptr<PixelYcbcr>> cudaQuantize(std::vector<std::shared_ptr<PixelYcbcr>> pixels, int block_size, bool all);
std::vector<std::shared_ptr<PixelYcbcr>> cudaUnquantize(std::vector<std::shared_ptr<PixelYcbcr>> pixels, int block_size, bool all);

//cuda dpcm.h
void cudaDPCM(std::vector<std::vector<std::shared_ptr<PixelYcbcr>>> blocks);
void cudaUnDPCM(std::vector<std::vector<std::shared_ptr<PixelYcbcr>>> blocks);

//cuda rle.h
std::shared_ptr<EncodedBlock> cudaRLE( std::vector<std::shared_ptr<PixelYcbcr>> block, int block_size);
std::vector<std::shared_ptr<PixelYcbcr>> cudaDecodeRLE( std::shared_ptr<EncodedBlock> encoded, int block_size);
