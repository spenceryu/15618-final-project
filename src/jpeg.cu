#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "dct.h"
#include "quantize.h"
#include "dpcm.h"
#include "rle.h"

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);

inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
                line);
        if(abort) {
            exit(code);
        }
    }
}

#else
#define cudaCheckError(ans) ans
#endif

//cuda image.h
// bytes is an array
CudaImageRgb* cudaConvertBytesToImage(unsigned char* bytes, unsigned int width, unsigned int height) {
    return NULL;
}
unsigned char* cudaConvertImageToBytes(CudaImageRgb* image) {
    return NULL;
}

CudaImageYcbcr* cudaConvertRgbToYcbcr(CudaImageRgb* input) {
    return NULL;
}
CudaImageRgb* cudaConvertYcbcrToRgb(CudaImageYcbcr* input) {
    return NULL;
}

CudaImageBlocks* cudaConvertYcbcrToBlocks(CudaImageYcbcr* input, int block_size) {
    return NULL;
}
CudaImageYcbcr* cudaConvertBlocksToYcbcr(CudaImageBlocks* input, int block_size) {
    return NULL;
}

//cuda dct.h
// returns array of pointers
// pixels is an array of pointers
PixelYcbcr** cudaDCT(PixelYcbcr** pixels, int block_size, bool all) {
    return NULL;
}
// returns array of pointers
// pixels is an array of pointers
PixelYcbcr** cudaIDCT(PixelYcbcr** pixels, int block_size, bool all) {
    return NULL;
}

//cuda quantize.h
// returns array of pointers
// pixels is an array of pointers
PixelYcbcr** cudaQuantize(PixelYcbcr** pixels, int block_size, bool all) {
    return NULL;
}
// returns array of pointers
// pixels is an array of pointers
PixelYcbcr** cudaUnquantize(PixelYcbcr** pixels, int block_size, bool all) {
    return NULL;
}

//cuda dpcm.h
void cudaDPCM(PixelYcbcr*** blocks) {
    return;
}
void cudaUnDPCM(PixelYcbcr*** blocks) {
    return;
}

//cuda rle.h
// block is an array of pointers
EncodedBlock* cudaRLE(PixelYcbcr** block, int block_size) {
    return NULL;
}
// returns an array of pointers
PixelYcbcr** cudaDecodeRLE(EncodedBlock* encoded, int block_size) {
    return NULL;
}
