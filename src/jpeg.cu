#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "dct.h"
#include "quantize.h"
#include "dpcm.h"
#include "rle.h"

#ifndef MACROBLOCK_SIZE
#define MACROBLOCK_SIZE 8
#endif

#define CUDA_THREADS_PER_BLOCK 64

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);

inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
                line);
        if(abort) {
            exit(code);
        }
    }
}

#else
#define cudaCheckError(ans) ans
#endif

// Kernels
__global__ void cudaDctKernel(CudaImageBlocks* imageBlocks, int block_size, bool all, CudaImageBlocks* device_result);

//cuda image.h
// bytes is an array
CudaImageRgb* cudaConvertBytesToImage(unsigned char* bytes, unsigned int width, unsigned int height) {
    return NULL;
}
unsigned char* cudaConvertImageToBytes(CudaImageRgb* image) {
    return NULL;
}

CudaImageYcbcr* cudaConvertRgbToYcbcr(CudaImageRgb* input) {
    return NULL;
}
CudaImageRgb* cudaConvertYcbcrToRgb(CudaImageYcbcr* input) {
    return NULL;
}

CudaImageBlocks* cudaConvertYcbcrToBlocks(CudaImageYcbcr* input, int block_size) {
    return NULL;
}
CudaImageYcbcr* cudaConvertBlocksToYcbcr(CudaImageBlocks* input, int block_size) {
    return NULL;
}

//cuda dct.h
// returns vector of blocks of pointers
void cudaDCT(CudaImageBlocks* imageBlocks, int block_size, bool all, CudaImageBlocks* device_result) {
    const int threadsPerBlock = CUDA_THREADS_PER_BLOCK;
    const int blocks = imageBlocks->numBlocks;
    cudaDctKernel<<<blocks, threadsPerBlock>>>(imageBlocks, block_size, all, device_result);
}

__global__ void cudaDctKernel(CudaImageBlocks* imageBlocks, int block_size, bool all, CudaImageBlocks* device_result) {
    PixelYcbcr** pixels = imageBlocks->blocks[blockIdx.x];
    PixelYcbcr** F = device_result->blocks[blockIdx.x];

    // Output: F(p, q)
    for (int p = 0; p < block_size; p++) {
        for (int q = 0; q < block_size; q++) {
            int vectorized_idx_pq = sub2ind(block_size, q, p);
            double ap = (p > 0) ? (sqrt(2/block_size)): (1/sqrt(block_size));
            double aq = (q > 0) ? (sqrt(2/block_size)) : (1/sqrt(block_size));

            double tmp_y, tmp_cb, tmp_cr;
            tmp_y = 0;
            tmp_cb = 0;
            tmp_cr = 0;

            // Input: f(m, n)
            for (int m = 0; m < block_size; m++) { // row
                for (int n = 0; n < block_size; n++) { // cow
                    int vectorized_idx_mn = sub2ind(block_size, n, m);
                    PixelYcbcr* f_mn = pixels[vectorized_idx_mn];

                    double xprod = cos((2*m + 1)*p*M_PI/(2*block_size));
                    double yprod = cos((2*n + 1)*q*M_PI/(2*block_size));

                    tmp_y += ((f_mn->y) * xprod * yprod);
                    if (all) {
                        tmp_cr += ((f_mn->cr) * xprod * yprod);
                        tmp_cb += ((f_mn->cb) * xprod * yprod);
                    }
                }
            }

            //F[vectorized_idx_pq] = std::make_shared<PixelYcbcr>();
            F[vectorized_idx_pq]->y = ap * aq * tmp_y;
            if (all) {
                F[vectorized_idx_pq]->cr = ap * aq * tmp_cr;
                F[vectorized_idx_pq]->cb = ap * aq * tmp_cb;
            } else {
                F[vectorized_idx_pq]->cr = 0;
                F[vectorized_idx_pq]->cb = 0;
            }
        }
    }
}

// returns vector of blocks of pointers
void cudaIDCT(CudaImageBlocks* imageBlocks, int block_size, bool all, CudaImageBlocks* device_result) {

}

//cuda quantize.h
void cudaQuantize(CudaImageBlocks* imageBlocks, int block_size, bool all, CudaImageBlocks* device_result) {

}
void cudaUnquantize(CudaImageBlocks* imageBlocks, int block_size, bool all, CudaImageBlocks* device_result) {

}

//cuda dpcm.h
void cudaDPCM(CudaImageBlocks* imageBlocks) {
    return;
}
void cudaUnDPCM(CudaImageBlocks* imageBlocks) {
    return;
}

//cuda rle.h
// block is an array of pointers
void cudaRLE(CudaImageBlocks* block, int block_size, EncodedBlock** device_result) {

}
// returns an array of pointers
void cudaDecodeRLE(EncodedBlock** encoded, int block_size, CudaImageBlocks* device_result) {

}

unsigned char* encodeParCuda(unsigned char* bytes, unsigned int width, unsigned int height) {

    //the pixels are now in the vector "image", 4 bytes per pixel, ordered RGBARGBA..., use it as texture, draw it, ...
    fprintf(stdout, "convertBytesToImage()...\n");
    CudaImageRgb* imageRgb = cudaConvertBytesToImage((unsigned char*) NULL, width, height);

    fprintf(stdout, "convertRgbToYcbcr()...\n");
    CudaImageYcbcr* imageYcbcr = cudaConvertRgbToYcbcr(imageRgb);

    fprintf(stdout, "convertYcbcrToBlocks()...\n");
    CudaImageBlocks* imageBlocks = cudaConvertYcbcrToBlocks(imageYcbcr, MACROBLOCK_SIZE);
    width = imageBlocks->width;
    height = imageBlocks->height;

    fprintf(stdout, "DCT()...\n");
    CudaImageBlocks* dcts = new CudaImageBlocks();
    dcts->width = imageBlocks->width;
    dcts->height = imageBlocks->height;
    cudaCheckError(hipMalloc(&dcts, imageBlocks->numBlocks));
    cudaDCT(imageBlocks, MACROBLOCK_SIZE, true, dcts);

    fprintf(stdout, "quantize()...\n");
    CudaImageBlocks* quantizedBlocks = new CudaImageBlocks();
    quantizedBlocks->width = imageBlocks->width;
    quantizedBlocks->height = imageBlocks->height;
    cudaCheckError(hipMalloc(&quantizedBlocks, imageBlocks->numBlocks));
    cudaQuantize(dcts, MACROBLOCK_SIZE, true, quantizedBlocks);

    fprintf(stdout, "DPCM()...\n");
    cudaDPCM(quantizedBlocks);

    fprintf(stdout, "RLE()...\n");
    // array of pointers
    EncodedBlock** encodedBlocks;
    cudaCheckError(hipMalloc(&encodedBlocks, imageBlocks->numBlocks));

    //TODO copy back to host

    fprintf(stdout, "warning: jpeg not being written, copy back to host\n");
    /*
    fprintf(stdout, "done encoding!\n");
    fprintf(stdout, "writing to file...\n");
    std::ofstream jpegFile(compressedFile);
    // TODO make this work
    for (int i = 0; i < imageBlocks->numBlocks; i++) {
        jpegFile << encodedBlocks[i];
    }
    fprintf(stdout, "jpeg stored!\n");
    */
    fprintf(stdout, "==============\n");
    fprintf(stdout, "now let's undo the process...\n");
    //TODO copy back to device

    fprintf(stdout, "undoing RLE()...\n");
    // array of array of pointers
    CudaImageBlocks* decodedQuantizedBlocks;
    cudaCheckError(hipMalloc(&decodedQuantizedBlocks, imageBlocks->numBlocks));
    cudaDecodeRLE(encodedBlocks, MACROBLOCK_SIZE, decodedQuantizedBlocks);

    fprintf(stdout, "undoing DPCM()...\n");
    cudaUnDPCM(decodedQuantizedBlocks);

    fprintf(stdout, "undoing quantize()...\n");
    // array of array of pointers
    PixelYcbcr*** unquantizedBlocks;
    unquantizedBlocks = (PixelYcbcr***) calloc(imageBlocks->numBlocks, sizeof(PixelYcbcr**));
    for (int i = 0; i < imageBlocks->numBlocks; i++) {
        unquantizedBlocks[i] = cudaUnquantize(decodedQuantizedBlocks[i], MACROBLOCK_SIZE, true);
    }

    fprintf(stdout, "undoing DCT()...\n");
    // array of array of pointers
    PixelYcbcr*** idcts;
    idcts = (PixelYcbcr***) calloc(imageBlocks->numBlocks, sizeof(PixelYcbcr**));
    for (int i = 0; i < imageBlocks->numBlocks; i++) {
        idcts[i] = cudaIDCT(unquantizedBlocks[i], MACROBLOCK_SIZE, true);
    }

    fprintf(stdout, "undoing convertYcbcrToBlocks()...\n");
    CudaImageBlocks imageBlocksIdct;
    imageBlocksIdct.blocks = idcts;
    imageBlocksIdct.width = width;
    imageBlocksIdct.height = height;
    CudaImageYcbcr* imgFromBlocks = cudaConvertBlocksToYcbcr(&imageBlocksIdct, MACROBLOCK_SIZE);

    fprintf(stdout, "undoing convertRgbToYcbcr()...\n");
    CudaImageRgb* imageRgbRecovered = cudaConvertYcbcrToRgb(imgFromBlocks);

    fprintf(stdout, "undoing convertBytesToImage()...\n");
    // array
    unsigned char* imgRecovered = cudaConvertImageToBytes(imageRgbRecovered);
    //TODO copy back to host
    return imgRecovered;
}
